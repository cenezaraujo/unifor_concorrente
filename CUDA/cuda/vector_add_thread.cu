#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define ERR 1e-6
__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    for(int i = index; i < n; i += stride){
        out[i] = a[i] + b[i];
    }
}
void verify(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        assert(fabs(out[i] - a[i] - b[i]) < ERR);
    }
    printf("OK\n");
}
int main(int argc, char *argv[]){
	if(argc<2) return 0;
	long N = atol(argv[1]);

    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){ a[i] = i; b[i] = i; }

    // Allocate device memory 
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    vector_add<<<1,256>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

	verify(out, a, b, N);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}

